#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <iostream>
#include <string>

/************************* NOTES *****************************

- The two input polynomials must have same degree, namely n-1
- The integer n must be a power of 2

**************************************************************/ 

using namespace std;

/*********** CUDA Helper functions from examples *******************/

struct cuda_exception {
    explicit cuda_exception(const char *err) : error_info(err) {}
    explicit cuda_exception(const string &err) : error_info(err) {}
    string what() const throw() { return error_info; }

    private:
    string error_info;
};

void checkCudaError(const char *msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        string error_info(msg);
        error_info += " : ";
        error_info += hipGetErrorString(err);
        throw cuda_exception(error_info);
    }
}

/*******************************************************************/

__global__ void poly_mult_ker(int *M, int n, int p) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    int i = id % n;
    int j = id / n;
    int d = i + j;
    M[(2 * n) + (d * n) + i] = (M[i] * M[n + j]) % p;
}

__global__ void reduce_terms_ker(int *M, int n, int p) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	for (int k = 1; k < n; k *= 2) {
		if ( id % (2*k) == 0)
			M[(2*n) + id] = (M[(2*n) + id] + M[(2*n) + id + k]) % p;
		// Sync to ensure values are ready for this step
		__syncthreads();
	}

	// Sync to ensure the result is ready
	__syncthreads();
	if ( id % n == 0)
		M[(2*n) + (2*n - 1) * n + (id / n)] = (M[(2*n) + (2*n - 1) * n + (id / n)] + M[(2*n) + id]) % p;

	for (int k = 1; k < n; k *= 2) {
		if ( id % (2*k) == 0 && id < ( n * (n-1)))
			M[(2*n) + (n*n) + id] = (M[(2*n) + (n*n) + id] + M[(2*n) + (n*n) + id + k]) % p;
		// Sync to ensure values are ready for this step
		__syncthreads();
	}

	// Sync to ensure the result is ready
	__syncthreads();
	if ( id % n == 0 && id < ( n * (n-1)) )
		M[(2*n) + (2*n - 1) * n + n + (id / n)] = (M[(2*n) + (2*n - 1) * n + n + (id / n)] + M[(2*n) + (n*n) + id]) % p;

}

__global__ void reduce_terms_ker_q2(int *M, int n, int p) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	for (int k = 1; k < n; k *= 2) {
		if ( id % (2*k) == 0)
			M[(2*n) + id] = (M[(2*n) + id] + M[(2*n) + id + k]) % p;
		// Sync to ensure values are ready for this step
		__syncthreads();
	}
	// Sync to ensure the result is ready
	__syncthreads();
	if ( id % n == 0)
		M[(2*n) + (2*n - 1) * n + (id / n)] = (M[(2*n) + (2*n - 1) * n + (id / n)] + M[(2*n) + id]) % p;

	for (int k = 1; k < n; k *= 2) {
		if ( id % (2*k) == 0 && id < ( n * (n-1)))
			M[(2*n) + (n*n) + id] = (M[(2*n) + (n*n) + id] + M[(2*n) + (n*n) + id + k]) % p;
		// Sync to ensure values are ready for this step
		__syncthreads();
	}

	// Sync to ensure the result is ready
	__syncthreads();
	if ( id % n == 0 && id < ( n * (n-1)) )
		M[(2*n) + (2*n - 1) * n + n + (id / n)] = (M[(2*n) + (2*n - 1) * n + n + (id / n)] + M[(2*n) + (n*n) + id]) % p;
}

void random_polynomials(int *M, size_t n, int p) {
	for ( int i = 0; i < 2*n; i++ ){
		int num = (int) rand() % p;
		M[i] = num;
    }
}

// Run kernel with coefficients of all 1 for simple verification
int poly_mult_test(int n_terms, int modulo_p, int question_id, int n_b, int n_t) {

    const int n = n_terms;
    const int p = modulo_p;

    // size n for a, size n for b, size (2*n-1) * n for coefficients of each term, size 2*n-1 for final summed coefficients
    const int worksp_size = (2 * n + ((2 * n - 1) * n) + (2*n)-1 );

    // Set coefficients to be 1 to easily verify the result
    int M[worksp_size] = {0};
    for (int i = 0; i < 2*n; i++) {
    	M[i] = 1;
    }

    // Display input polynomials
    printf("\n============== INPUT & RESULTS ==========\n\n");
    printf("Input - Polynomial A Coefficients:\n");
    for (int i=0; i<n; i++) {
		printf("%d ", M[i]);
	}
	printf("\n\nInput - Polynomial B Coefficients:\n");
	for (int i=0; i<n; i++) {
		printf("%d ", M[i+n]);
	}
	printf("\n\n");

	// Allocate GPU memory for the workspace
    int *Md;
    hipMalloc((void **)&Md, sizeof(int)*worksp_size);
    checkCudaError("allocate GPU memory for the workspace");
    hipMemcpy(Md, M, sizeof(int)*worksp_size, hipMemcpyHostToDevice);

    poly_mult_ker<<<n_b, n_t>>>(Md, n, p);

    if (question_id == 1) {
	    reduce_terms_ker<<<n_b, n_t>>>(Md, n, p);
	} else if (question_id == 2) {
	    reduce_terms_ker_q2<<<n_b, n_t>>>(Md, n, p);
	}

	// Copy GPU memory for the workspace back to host
    hipMemcpy(M, Md, sizeof(int)*worksp_size, hipMemcpyDeviceToHost);	

    // Display resulting polynomial
    int result_start = 2 * n + ((2 * n - 1) * n);
    int result_length = (2 * n) - 1;
    printf("Result - Polynomial A*B Coefficients:\n");
    for (int i = result_start; i < result_start + result_length; i++)
    	printf("%d ", M[i]);
    printf("\n");

    int isCorrect = 1;
    for (int i = result_start; i < result_start + result_length - 1; i++) {
    	int diff = abs(M[i] - M[i+1]);
    	if ( !(diff == 1 || diff == (p - 1)) ) isCorrect = 0;
    }

    if (isCorrect) {
    	printf("\nResult is correct!\n");
    } else {
    	printf("\nResult is INCORRECT!\n");
    }

    hipFree(Md);

    return 0;
}

// Run kernel with random polynomial input
int poly_mult(int n_terms, int modulo_p, int q, int n_b, int n_t, int is_dev_mode) {
    const int n = n_terms;
    const int p = modulo_p;

    // size n for a, size n for b, size (2*n-1) * n for coefficients of each term, size 2*n-1 for final summed coefficients
    const int worksp_size = (2 * n + ((2 * n - 1) * n) + (2*n)-1 );
    int M[worksp_size] = {0};
    random_polynomials(M, n, p);

    // Display input polynomials
    printf("\n============== INPUT & RESULTS ==========\n\n");
    printf("Input - Polynomial A Coefficients:\n");
    for (int i=0; i<n; i++) {
		printf("%d ", M[i]);
	}
	printf("\n\nInput - Polynomial B Coefficients:\n");
	for (int i=0; i<n; i++) {
		printf("%d ", M[i+n]);
	}
	printf("\n\n");

	// Allocate GPU memory for the workspace
    int *Md;
    hipMalloc((void **)&Md, sizeof(int)*worksp_size);
    checkCudaError("allocate GPU memory for the workspace");
    hipMemcpy(Md, M, sizeof(int)*worksp_size, hipMemcpyHostToDevice);

    poly_mult_ker<<<n_b, n_t>>>(Md, n, p);

    if (q == 1) {
	    reduce_terms_ker<<<n_b, n_t>>>(Md, n, p);
	} else if (q == 2) {
	    reduce_terms_ker_q2<<<n_b, n_t>>>(Md, n, p);
	}

	// Copy GPU memory for the workspace back to host
    hipMemcpy(M, Md, sizeof(int)*worksp_size, hipMemcpyDeviceToHost);

    // Display workspace values if dev mode is true
    if (is_dev_mode) {
	    //Debug workspace
	    printf("\nDEBUG WORKSPACE:\n");
	    for (int i = 2*n; i < worksp_size; ++i) {
	    	if (i % n == 0)
	    		printf(" . ");
	    	printf("%d ", M[i]);
	    }
	    printf("\n\n\n");	
    }

    // Display resulting polynomial
    int result_start = 2 * n + ((2 * n - 1) * n);
    int result_length = (2 * n) - 1;
    printf("Result - Polynomial A*B Coefficients:\n");
    for (int i = result_start; i < result_start + result_length; i++)
    	printf("%d ", M[i]);
    printf("\n");

    hipFree(Md);

    return 0;
}

void print_usage() {
	printf("Usage: ./poly_mult [MODE: run, dev, test] [QUESTION ID: (integer) 1 OR 2] [ N_TERMS: integer power of 2 ] [ MODULO_P: integer small prime (e.g. 103) ]\n");
}

int validate_args(int argc, char **argv) {
	if ( argc < 3 )
		return 1;

	string mode = argv[1];
	int question_id = atoi(argv[2]);

	if ( !(mode.compare("run") == 0) && !(mode.compare("dev") == 0) && !(mode.compare("test") == 0) ) {
		printf("Invalid Mode!\n");
		return 1;
	}

	if ( ! (question_id == 1 || question_id == 2) ) {
		printf("Invalid Question Id!\n");
		return 1;
	}

	if ( argc < 5 ) {
		printf("Please provide values for both: [N_TERMS] [MODULO_P]\n");
		return 1;
	}

	return 0;
}

int main(int argc, char **argv) {

	if (validate_args(argc, argv)) {
		print_usage();
		exit(1);
	}

	string mode = argv[1];
	int question_id = atoi(argv[2]);
	int n_terms = atoi(argv[3]);
	int modulo_p = atoi(argv[4]);
	int n_blocks, n_threads;

	if (question_id == 1) {
		n_threads = n_terms;
		n_blocks = n_terms;
	} else if (question_id == 2) {
		n_threads = 64;
		n_blocks = (n_terms*n_terms) / n_threads;
	}

	printf("\n======= ARGUMENTS =======\n");
	cout << "MODE: " << argv[1] << "\n";
	printf("question_id: %d\n", question_id);
	printf("n_terms: %d\n", n_terms);
	printf("modulo_p: %d\n", modulo_p);
	printf("n_blocks: %d\n", n_blocks);
	printf("n_threads: %d\n", n_threads);
	printf("=========================\n");


	if ( mode.compare("run") == 0 ) {
		poly_mult(n_terms, modulo_p, question_id, n_blocks, n_threads, 0);
	}

	if ( mode.compare("dev") == 0 ) {
		poly_mult(n_terms, modulo_p, question_id, n_blocks, n_threads, 1);
	}

	if ( mode.compare("test") == 0 ) {
		poly_mult_test(n_terms, modulo_p, question_id, n_blocks, n_threads);
	}

	return 0;
}